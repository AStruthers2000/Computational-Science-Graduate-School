//
// Created by Strut on 5/6/2024.
//
#include "RandomNumberGenerator.cuh"
#include <iostream>

//defines for error checking and handling
#define CUDA_CALL(x) do { hipError_t err = (x); if(err != hipSuccess){ \
    printf("Error %d at: %s:%d\n\t%s\n", err, __FILE__, __LINE__, hipGetErrorString(err)); \
    exit(EXIT_FAILURE);}} while(0)

#define CURAND_CALL(x) do { hiprandStatus_t err = (x); if(err !=HIPRAND_STATUS_SUCCESS) { \
    printf("Error %d at %s:%d\n",err, __FILE__,__LINE__);\
    exit(EXIT_FAILURE);}} while(0)

/** \brief Builds a CuRAND generator from some given initial parameters.
 * Useful for configuring random number generators neatly
 * @param out_gen An out parameter that will be the configured generator upon success
 * @param rng_type The RNG type desired for this generator
 * @param seed An integer that seeds the given generator. Default = 0 means a seed will be randomly generated
 * @param offset An offset used to skip portions of the RNG cycle, effectively a random starting point. Default = 0 means an offset will be randomly generated
 */


__host__ void
build_curand_generator(hiprandGenerator_t &out_gen, hiprandRngType_t rng_type, int seed, unsigned int offset)
{
    CURAND_CALL(hiprandCreateGenerator(&out_gen, rng_type));

    if(seed == 0) seed = static_cast<int>(time(nullptr));
    CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(out_gen, seed));

    hiprandOrdering_t order;
    switch(rng_type)
    {
        case HIPRAND_RNG_PSEUDO_XORWOW:
            order = HIPRAND_ORDERING_PSEUDO_SEEDED;
            break;
        case HIPRAND_RNG_PSEUDO_MRG32K3A:
            order = HIPRAND_ORDERING_PSEUDO_DYNAMIC;
            break;
        case HIPRAND_RNG_PSEUDO_MTGP32:
            //order = HIPRAND_ORDERING_PSEUDO_BEST;
            //break;
        case HIPRAND_RNG_PSEUDO_MT19937:
            order = HIPRAND_ORDERING_PSEUDO_BEST;
            break;
        case HIPRAND_RNG_PSEUDO_PHILOX4_32_10:
            order = HIPRAND_ORDERING_PSEUDO_DYNAMIC;
            break;
        default:
            order = HIPRAND_ORDERING_PSEUDO_DEFAULT;
            break;
    }
    CURAND_CALL(hiprandSetGeneratorOrdering(out_gen, order));

    if(rng_type != HIPRAND_RNG_PSEUDO_MTGP32 && rng_type != HIPRAND_RNG_PSEUDO_MT19937) {
        if (offset == 0) {
            unsigned int *random_offset;
            CUDA_CALL(hipMallocManaged(&random_offset, sizeof(unsigned int)));
            CURAND_CALL(hiprandGenerate(out_gen, random_offset, 1));
            CUDA_CALL(hipDeviceSynchronize());
            offset = *random_offset;
            CUDA_CALL(hipFree(random_offset));
        }

        CURAND_CALL(hiprandSetGeneratorOffset(out_gen, offset));
    }
}

__host__ void generate_random_numbers(hiprandGenerator_t generator, long double *outputPtr, size_t num, float lower, float upper)
{
    float *dev_numbers, *host_numbers;
    host_numbers = (float*)(malloc(sizeof(float) * num));
    CUDA_CALL(hipMalloc(&dev_numbers, sizeof(float) * num));
    CURAND_CALL(hiprandGenerateUniform(generator, dev_numbers, num));
    CUDA_CALL(hipDeviceSynchronize());
    hipMemcpy(host_numbers, dev_numbers, sizeof(float) * num, hipMemcpyDeviceToHost);
    for(int i = 0; i < num; i++)
    {
        auto n = static_cast<long double>(host_numbers[i]);
        outputPtr[i] = lower + (n * (upper - lower));
    }
    CUDA_CALL(hipFree(dev_numbers));
    free(host_numbers);
}

